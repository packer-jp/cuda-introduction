#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h> 
#include <hip/hip_runtime.h>
/*

さらなる高速化を目指して

*/

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e = hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

/*

行列積の場合

for (int i = 0; i < nx; ++i){
    for (int j = 0; j < ny; ++j){
        // iとjを用いた処理
    }
}

--------CUDA化--------

int i = threadIdx.y + blockDim.y * blockIdx.y; <- xかyかはiとjの使われ方による
int j = threadIdx.x + blockDim.x * blockIdx.x;
//iとjを用いた処理

インプレンス社: CUDA Cのp59とかにあった
*/

void initMatrix(double *mat, double init_num, int dim){
    // mat is expected to be a 2-dimentional matrix expressed by a 1-dimentional array. 
    // each dimention of mat is expected to be the same.
    for (int i = 0; i < dim; ++i) {
        for (int j = 0; j < dim; ++j){
            mat[i*dim+j] = init_num;
        }
    }
}


double calculateElapsedTime(struct timespec start_time, struct timespec end_time){
    return (double) (end_time.tv_sec - start_time.tv_sec) + (double) (end_time.tv_nsec - start_time.tv_nsec) * 1e-9;
}

__global__ void matMulGpu(double *input_mat1, double *input_mat2, double *output_mat, int dim){
    // mat is expected to be a 2-dimentional matrix expressed by a 1-dimentional array. 
    // each dimention of mat is expected to be the same.
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int k = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (i >= dim || j >= dim || k >= dim) return;
    atomicAdd(&output_mat[i*dim+j], input_mat1[i*dim+k] * input_mat2[k*dim+j]); // 競合状態解消のため
}

void terminate(const char *error_sentence){
    perror(error_sentence);
    exit(1);
}

void debugMatrix(double *mat, int dim){
    for (int i = 0; i < dim; ++i){
        for (int j = 0; j < dim; ++j){
            printf("%f ", mat[i*dim+j]);
        }
        printf("\n");
    }
}

int main(int argc, char **argv){
    if (argc < 2){
        terminate("Usage matmul_gpu dim_size");
    }

    //device set up
    int device_id = 0;
    hipSetDevice(device_id);

    int n = atoi(argv[1]);
    struct timespec start_time, end_time;
    double *input_mat1 = (double *)malloc(sizeof(double)*n*n);
    double *input_mat2 = (double *)malloc(sizeof(double)*n*n);
    double *output_mat = (double *)malloc(sizeof(double)*n*n);

    initMatrix(input_mat1, 3.0, n);
    initMatrix(input_mat2, 0.1, n);
    initMatrix(output_mat, 0.0, n);

    double *d_input_mat1, *d_input_mat2, *d_output_mat;
    hipMalloc((void**) &d_input_mat1,sizeof(double)*n*n);
    hipMalloc((void**) &d_input_mat2,sizeof(double)*n*n);
    hipMalloc((void**) &d_output_mat,sizeof(double)*n*n);

    hipMemcpy(d_input_mat1, input_mat1, sizeof(double)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_input_mat2, input_mat2, sizeof(double)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_output_mat, output_mat, sizeof(double)*n*n, hipMemcpyHostToDevice);

    int num_threads = 8; // warpのせいか、ここが32の倍数じゃないので遅くなります。
    assert(num_threads * num_threads < 64 + 1);

    dim3 block(num_threads, num_threads, num_threads);
    dim3 grid((n+block.x-1)/block.x, (n+block.y-1)/block.y, (n+block.z-1)/block.z);

    clock_gettime(CLOCK_REALTIME, &start_time);
    matMulGpu<<<grid, block>>>(d_input_mat1, d_input_mat2, d_output_mat, n);
    hipDeviceSynchronize(); // Wait until GPU processing finishs.
    cudaCheckError();
    hipMemcpy(output_mat, d_output_mat, sizeof(double)*n*n, hipMemcpyDeviceToHost); 
    clock_gettime(CLOCK_REALTIME, &end_time);
    hipFree(d_input_mat1);
    hipFree(d_input_mat2);
    hipFree(d_output_mat);

    //debugMatrix(output_mat, n); 
    printf("elapsed time %f\n", calculateElapsedTime(start_time, end_time));

    return 0;
}